#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava, 2020/11
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage with unified memory.
//
// Image transformation from RGB to BW schema.
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include "font24x40_lsb.h"
#define FONT_W 24
#define FONT_H 40

#include "cuda_img.h"
#include "animation.h"

// Demo kernel to transform RGB color schema to BW schema
__global__ void kernel_grayscale(CudaImg t_color_cuda_img, CudaImg t_bw_cuda_img)
{
    // X,Y coordinates and check image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if (l_y >= t_color_cuda_img.m_size.y)
        return;
    if (l_x >= t_color_cuda_img.m_size.x)
        return;

    // Get point from color picture
    uchar3 l_bgr = t_color_cuda_img.m_p_uchar3[l_y * t_color_cuda_img.m_size.x + l_x];

    // Store BW point to new image
    t_bw_cuda_img.m_p_uchar1[l_y * t_bw_cuda_img.m_size.x + l_x].x = l_bgr.x * 0.11 + l_bgr.y * 0.59 + l_bgr.z * 0.30;
}

void cu_run_grayscale(CudaImg t_color_cuda_img, CudaImg t_bw_cuda_img)
{
    hipError_t l_cerr;

    // Grid creation, size of grid must be equal or greater than images
    int l_block_size = 16;
    dim3 l_blocks((t_color_cuda_img.m_size.x + l_block_size - 1) / l_block_size, (t_color_cuda_img.m_size.y + l_block_size - 1) / l_block_size);
    dim3 l_threads(l_block_size, l_block_size);
    kernel_grayscale<<<l_blocks, l_threads>>>(t_color_cuda_img, t_bw_cuda_img);

    if ((l_cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    hipDeviceSynchronize();
}

__global__ void kernel_insert_image(CudaImg t_cuda_big_img, CudaImg t_cuda_small_img, int2 t_pos, uchar3 mask)
{
    // X,Y coordinates and check image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if (l_y >= t_cuda_small_img.m_size.y || l_y + t_pos.y >= t_cuda_big_img.m_size.y)
        return;
    if (l_x >= t_cuda_small_img.m_size.x || l_x + t_pos.x >= t_cuda_big_img.m_size.x)
        return;

    int small_index = l_y * t_cuda_small_img.m_size.x + l_x;
    int big_index = (l_y + t_pos.y) * t_cuda_big_img.m_size.x + (l_x + t_pos.x);

    // Get point from color picture
    uchar4 fg = t_cuda_small_img.m_p_uchar4[small_index];
    uchar3 bg = t_cuda_big_img.m_p_uchar3[big_index];

    // l_bgr.x *= mask.x;
    // l_bgr.y *= mask.y;
    // l_bgr.z *= mask.z;

    float alpha = fg.w / 255.0f;

    uchar3 out;
    out.x = (alpha * fg.x + (1 - alpha) * bg.x);
    out.y = (alpha * fg.y + (1 - alpha) * bg.y);
    out.z = (alpha * fg.z + (1 - alpha) * bg.z);

    // Store point at position in big image
    // t_cuda_big_img.m_p_uchar3[(t_pos.y + l_y) * t_cuda_big_img.m_size.x + (t_pos.x + l_x)] = l_bgr;
    t_cuda_big_img.at3(l_y + t_pos.y, l_x + t_pos.x, out);
}

void cu_insert_image(CudaImg &t_cuda_big_img, CudaImg &t_cuda_small_img, int2 t_pos, uchar3 mask)
{
    hipError_t l_cerr;

    int l_block_size = 16;
    dim3 l_blocks((t_cuda_small_img.m_size.x + l_block_size - 1) / l_block_size, (t_cuda_small_img.m_size.y + l_block_size - 1) / l_block_size);
    dim3 l_threads(l_block_size, l_block_size);
    kernel_insert_image<<<l_blocks, l_threads>>>(t_cuda_big_img, t_cuda_small_img, t_pos, mask);

    if ((l_cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    hipDeviceSynchronize();
}

__global__ void kernel_swap_image(CudaImg t_cuda_img1, CudaImg t_cuda_img2)
{
    // X,Y coordinates and check image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if (l_y >= t_cuda_img2.m_size.y || l_y >= t_cuda_img1.m_size.y)
        return;
    if (l_x >= t_cuda_img2.m_size.x || l_x >= t_cuda_img1.m_size.x)
        return;

    // Get point from color picture
    uchar3 l_bgr = t_cuda_img2.m_p_uchar3[l_y * t_cuda_img2.m_size.x + l_x];

    t_cuda_img2.at3(l_y, l_x, t_cuda_img1.m_p_uchar3[l_y * t_cuda_img1.m_size.x + l_x]);

    // Store point at position in big image
    t_cuda_img1.at3(l_y, l_x, l_bgr);
}

void cu_swap_image(CudaImg &t_cuda_img1, CudaImg &t_cuda_img2)
{
    hipError_t l_cerr;

    int l_block_size = 16;
    dim3 l_blocks((t_cuda_img2.m_size.x + l_block_size - 1) / l_block_size, (t_cuda_img2.m_size.y + l_block_size - 1) / l_block_size);
    dim3 l_threads(l_block_size, l_block_size);
    kernel_swap_image<<<l_blocks, l_threads>>>(t_cuda_img1, t_cuda_img2);

    if ((l_cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    hipDeviceSynchronize();
}

__global__ void kernel_swap2_image(CudaImg t_cuda_img1, CudaImg t_cuda_img2, int2 t_pos)
{
    // X,Y coordinates and check image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if (l_y >= t_cuda_img2.m_size.y || l_y + t_pos.y >= t_cuda_img1.m_size.y)
        return;
    if (l_x >= t_cuda_img2.m_size.x || l_x + t_pos.x >= t_cuda_img1.m_size.x)
        return;

    // Get point from color picture
    uchar3 l_bgr = t_cuda_img2.m_p_uchar3[l_y * t_cuda_img2.m_size.x + l_x];

    t_cuda_img2.at3(l_y, l_x, t_cuda_img1.m_p_uchar3[(l_y + t_pos.y) * t_cuda_img1.m_size.x + l_x + t_pos.x]);

    // Store point at position in big image
    t_cuda_img1.at3(l_y + t_pos.y, l_x + t_pos.x, l_bgr);
}

void cu_swap2_image(CudaImg &t_cuda_img1, CudaImg &t_cuda_img2, CudaImg &helper)
{
    hipError_t l_cerr;

    int2 pos1 = make_int2(0, 0);
    int2 pos2 = make_int2(helper.m_size.x, 0);
    int2 pos3 = make_int2(0, helper.m_size.y);
    int2 pos4 = make_int2(helper.m_size.x, helper.m_size.y);

    int l_block_size = 16;
    dim3 l_blocks((t_cuda_img2.m_size.x + l_block_size - 1) / l_block_size, (t_cuda_img2.m_size.y + l_block_size - 1) / l_block_size);
    dim3 l_threads(l_block_size, l_block_size);

    kernel_swap2_image<<<l_blocks, l_threads>>>(t_cuda_img1, helper, pos1);
    kernel_swap2_image<<<l_blocks, l_threads>>>(t_cuda_img2, helper, pos1);
    kernel_swap2_image<<<l_blocks, l_threads>>>(t_cuda_img1, helper, pos1);

    // kernel_swap2_image<<<l_blocks, l_threads>>>(t_cuda_img1, helper, pos2);
    // kernel_swap2_image<<<l_blocks, l_threads>>>(t_cuda_img2, helper, pos2);
    // kernel_swap2_image<<<l_blocks, l_threads>>>(t_cuda_img1, helper, pos2);

    // kernel_swap2_image<<<l_blocks, l_threads>>>(t_cuda_img1, helper, pos3);
    // kernel_swap2_image<<<l_blocks, l_threads>>>(t_cuda_img2, helper, pos3);
    // kernel_swap2_image<<<l_blocks, l_threads>>>(t_cuda_img1, helper, pos3);

    kernel_swap2_image<<<l_blocks, l_threads>>>(t_cuda_img1, helper, pos4);
    kernel_swap2_image<<<l_blocks, l_threads>>>(t_cuda_img2, helper, pos4);
    kernel_swap2_image<<<l_blocks, l_threads>>>(t_cuda_img1, helper, pos4);

    if ((l_cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    hipDeviceSynchronize();
}

__global__ void kernel_rotate90_image(CudaImg t_cu_img, CudaImg t_cu_img_rotated, int t_direction)
{
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;

    if (l_y >= t_cu_img.m_size.y || l_x >= t_cu_img.m_size.x)
        return;

    uchar4 pixel = t_cu_img.m_p_uchar4[l_y * t_cu_img.m_size.x + l_x];

    if (t_direction == 0)
    {
        // Rotate 90° clockwise
        t_cu_img_rotated.at4(l_x, t_cu_img.m_size.y - l_y - 1, pixel);
    }
    else
    {
        // Rotate 90° counterclockwise
        t_cu_img_rotated.at4(t_cu_img.m_size.x - l_x - 1, l_y, pixel);
    }
}

void cu_rotate90(CudaImg &t_cu_img, CudaImg &t_cu_img_rotated, int t_direction)
{
    hipError_t l_cerr;

    int l_block_size = 16;
    dim3 l_blocks((t_cu_img.m_size.x + l_block_size - 1) / l_block_size,
                  (t_cu_img.m_size.y + l_block_size - 1) / l_block_size);
    dim3 l_threads(l_block_size, l_block_size);
    kernel_rotate90_image<<<l_blocks, l_threads>>>(t_cu_img, t_cu_img_rotated, t_direction);

    if ((l_cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    hipDeviceSynchronize();
}

__global__ void kernel_scale_image(CudaImg t_cu_orig, CudaImg t_cu_scaled, float2 l_scale)
{
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;

    if (l_x >= t_cu_scaled.m_size.x || l_y >= t_cu_scaled.m_size.y)
        return;

    float l_orig_x = l_x * l_scale.x;
    float l_orig_y = l_y * l_scale.y;

    int x = (int)l_orig_x;
    int y = (int)l_orig_y;

    if (x >= t_cu_orig.m_size.x - 1 || y >= t_cu_orig.m_size.y - 1)
        return; // avoid overflow

    float l_diff_x = l_orig_x - x;
    float l_diff_y = l_orig_y - y;

    uchar4 bgr00 = t_cu_orig.m_p_uchar4[y * t_cu_orig.m_size.x + x];
    uchar4 bgr01 = t_cu_orig.m_p_uchar4[y * t_cu_orig.m_size.x + (1 + x)];
    uchar4 bgr10 = t_cu_orig.m_p_uchar4[(1 + y) * t_cu_orig.m_size.x + x];
    uchar4 bgr11 = t_cu_orig.m_p_uchar4[(1 + y) * t_cu_orig.m_size.x + (1 + x)];

    uchar4 bgr;

    bgr.x = bgr00.x * (1 - l_diff_y) * (1 - l_diff_x) +
            bgr01.x * (1 - l_diff_y) * (l_diff_x) +
            bgr10.x * (l_diff_y) * (1 - l_diff_x) +
            bgr11.x * (l_diff_y) * (l_diff_x);

    bgr.y = bgr00.y * (1 - l_diff_y) * (1 - l_diff_x) +
            bgr01.y * (1 - l_diff_y) * (l_diff_x) +
            bgr10.y * (l_diff_y) * (1 - l_diff_x) +
            bgr11.y * (l_diff_y) * (l_diff_x);

    bgr.z = bgr00.z * (1 - l_diff_y) * (1 - l_diff_x) +
            bgr01.z * (1 - l_diff_y) * (l_diff_x) +
            bgr10.z * (l_diff_y) * (1 - l_diff_x) +
            bgr11.z * (l_diff_y) * (l_diff_x);

    bgr.w = bgr00.w * (1 - l_diff_y) * (1 - l_diff_x) +
            bgr01.w * (1 - l_diff_y) * (l_diff_x) +
            bgr10.w * (l_diff_y) * (1 - l_diff_x) +
            bgr11.w * (l_diff_y) * (l_diff_x);

    t_cu_scaled.at4(l_y, l_x, bgr);
}

void cu_scale(CudaImg &t_cu_orig, CudaImg &t_cu_scaled)
{
    hipError_t l_cerr;

    float2 scale = make_float2(
        (float)(t_cu_orig.m_size.x - 1) / (float)t_cu_scaled.m_size.x,
        (float)(t_cu_orig.m_size.y - 1) / (float)t_cu_scaled.m_size.y);

    int l_block_size = 16;
    dim3 l_blocks((t_cu_scaled.m_size.x + l_block_size - 1) / l_block_size,
                  (t_cu_scaled.m_size.y + l_block_size - 1) / l_block_size);
    dim3 l_threads(l_block_size, l_block_size);
    kernel_scale_image<<<l_blocks, l_threads>>>(t_cu_orig, t_cu_scaled, scale);

    if ((l_cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    hipDeviceSynchronize();
}

__global__ void kernel_character_image(CudaImg t_cu_img, char t_char, int2 pos, uchar4 color)
{
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;

    if (l_y + pos.y >= t_cu_img.m_size.y || l_x + pos.x >= t_cu_img.m_size.x)
        return;
    if (l_y >= FONT_H || l_x >= FONT_W)
    {
        return;
    }

    if (font[(int)t_char][l_y] & (1 << l_x))
    {
        t_cu_img.at4(l_y + pos.y, l_x + pos.x, color);
    }
}

void cu_character(CudaImg t_cu_img, char t_char, int2 pos, uchar4 color)
{
    hipError_t l_cerr;

    int l_block_size = 16;
    dim3 l_blocks((FONT_W + l_block_size - 1) / l_block_size,
                  (FONT_H + l_block_size - 1) / l_block_size);
    dim3 l_threads(l_block_size, l_block_size);
    kernel_character_image<<<l_blocks, l_threads>>>(t_cu_img, t_char, pos, color);

    if ((l_cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    hipDeviceSynchronize();
}

__global__ void kernel_run_rotate(CudaImg t_cv_img_orig, CudaImg t_cv_img_rotate, float t_sin, float t_cos)
{
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if (l_y >= t_cv_img_rotate.m_size.y || l_x >= t_cv_img_rotate.m_size.x)
        return;

    // recalculation from image coordinates to centerpoint coordinates
    int l_crotate_x = l_x - t_cv_img_rotate.m_size.x / 2;
    int l_crotate_y = l_y - t_cv_img_rotate.m_size.y / 2;

    // position in orig image
    float l_corig_x = t_cos * l_crotate_x - t_sin * l_crotate_y;
    float l_corig_y = t_sin * l_crotate_x + t_cos * l_crotate_y;

    // recalculation from centerpoint coordinates to image coordinates
    int l_orig_x = l_corig_x + t_cv_img_orig.m_size.x / 2;
    int l_orig_y = l_corig_y + t_cv_img_orig.m_size.y / 2;

    // out of orig image?
    if (l_orig_x < 0 || l_orig_x >= t_cv_img_orig.m_size.x)
        return;
    if (l_orig_y < 0 || l_orig_y >= t_cv_img_orig.m_size.y)
        return;

    uchar4 point = t_cv_img_orig.m_p_uchar4[l_orig_y * t_cv_img_orig.m_size.x + l_orig_x];

    t_cv_img_rotate.at4(l_y, l_x, point);
}

void cu_run_rotate(CudaImg &t_cv_img_orig, CudaImg &t_cv_img_rotate, float t_angle)
{
    float t_sin = sinf(t_angle);
    float t_cos = cosf(t_angle);

    hipError_t l_cerr;

    int l_block_size = 16;
    dim3 l_blocks((t_cv_img_rotate.m_size.x + l_block_size - 1) / l_block_size,
                  (t_cv_img_rotate.m_size.y + l_block_size - 1) / l_block_size);
    dim3 l_threads(l_block_size, l_block_size);

    kernel_run_rotate<<<l_blocks, l_threads>>>(t_cv_img_orig, t_cv_img_rotate, t_sin, t_cos);

    if ((l_cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    hipDeviceSynchronize();
}

void Animation::start(CudaImg &t_bg_cuda_img, CudaImg &t_ins_cuda_img, CudaImg &t_rot_cuda_img)
{
    if (m_initialized)
        return;
    hipError_t l_cerr;

    m_bg_cuda_img = t_bg_cuda_img;   // background image
    m_res_cuda_img = t_bg_cuda_img;  // result image
    m_ins_cuda_img = t_ins_cuda_img; // insert image
    m_rot_cuda_img = t_rot_cuda_img;

    // Memory for background
    l_cerr = hipMalloc(&m_bg_cuda_img.m_p_void, m_bg_cuda_img.m_size.x * m_bg_cuda_img.m_size.y * sizeof(uchar3));
    if (l_cerr != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    // Memory for result
    l_cerr = hipMalloc(&m_res_cuda_img.m_p_void, m_res_cuda_img.m_size.x * m_res_cuda_img.m_size.y * sizeof(uchar3));
    if (l_cerr != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    // Memory for inserted image
    l_cerr = hipMalloc(&m_ins_cuda_img.m_p_void, m_ins_cuda_img.m_size.x * m_ins_cuda_img.m_size.y * sizeof(uchar4));
    if (l_cerr != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));
    
    // Memory for rotated image
    l_cerr = hipMalloc(&m_rot_cuda_img.m_p_void, m_rot_cuda_img.m_size.x * m_rot_cuda_img.m_size.y * sizeof(uchar4));
    if (l_cerr != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    // Copy data to GPU device
    l_cerr = hipMemcpy(m_ins_cuda_img.m_p_void, t_ins_cuda_img.m_p_void, m_ins_cuda_img.m_size.x * m_ins_cuda_img.m_size.y * sizeof(uchar4), hipMemcpyHostToDevice);
    if (l_cerr != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    m_initialized = 1;
}

void Animation::next(CudaImg &t_res_pic, float angle, int2 t_position, uchar3 t_mask)
{
    if (!m_initialized)
        return;

    hipError_t cerr;

    // Copy data internally GPU from background into result
    cerr = hipMemcpy(m_res_cuda_img.m_p_void, m_bg_cuda_img.m_p_void, m_bg_cuda_img.m_size.x * m_bg_cuda_img.m_size.y * sizeof(uchar3), hipMemcpyDeviceToDevice);
    if (cerr != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(cerr));

    // rotate image
    cu_run_rotate(m_ins_cuda_img, m_rot_cuda_img, angle);

    // insert image
    cu_insert_image(m_res_cuda_img, m_rot_cuda_img, t_position, t_mask);

    // Copy data to GPU device
    cerr = hipMemcpy(t_res_pic.m_p_void, m_res_cuda_img.m_p_void, m_res_cuda_img.m_size.x * m_res_cuda_img.m_size.y * sizeof(uchar3), hipMemcpyDeviceToHost);
    if (cerr != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(cerr));
}

void Animation::stop()
{
    if (!m_initialized)
        return;

    hipFree(m_bg_cuda_img.m_p_void);
    hipFree(m_res_cuda_img.m_p_void);
    hipFree(m_ins_cuda_img.m_p_void);

    m_initialized = 0;
}